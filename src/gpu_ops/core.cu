#include "hip/hip_runtime.h"
#include <stdio.h>
#include <gpu_ops/core.cuh>


__global__ void gpu_print() {
	printf("Hello, GPU-accelarated world!\n");
}


void test_gpu_print() {
	gpu_print<<<1, 1>>>();
}